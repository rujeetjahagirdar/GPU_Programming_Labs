#include "hip/hip_runtime.h"
#include "gputk.h"
#include<iostream>
#include "color_to_grayscale_gpu.h"
using namespace std;
__global__
void colorToGrayscale_kernel(float *output, float *input, int width, int height) {
    // TODO: Implement Kernel
    // TODO: Complete CUDA Kernel for image blurring.
     int channels = 3;
     int x = blockIdx.x * blockDim.x + threadIdx.x;
     int y = blockIdx.y * blockDim.y + threadIdx.y;
     if(x>=width || y>=height) return;
     float Values[3] = {0.0f, 0.0f, 0.0f};
     for (int c=0; c<channels; ++c) {
 	int i = y * width * channels + x * channels + c;
     	Values[c] = input[i];
     }
     output[y * width + x] = 0.299f * Values[0] + 0.587f * Values[1] + 0.114f * Values[2];

}

void colorToGrayscale(float *output, float *input, int width, int height) {
    float *deviceInputImageData = nullptr;
    float *deviceOutputImageData = nullptr;
    gpuTKTime_start(GPU, "Doing GPU Computation (memory + compute)");

    gpuTKTime_start(GPU, "Doing GPU memory allocation");
    // TODO: Allocate GPU memory here
    hipMalloc(&deviceInputImageData, 3 * width * height * sizeof(float));
    hipMalloc(&deviceOutputImageData,  width * height * sizeof(float));
    gpuTKTime_stop(GPU, "Doing GPU memory allocation");

    gpuTKTime_start(Copy, "Copying data to the GPU");
    // TODO: Copy data to GPU here
    hipMemcpy(deviceInputImageData, input, 3 * width * height * sizeof(float), hipMemcpyHostToDevice);
    gpuTKTime_stop(Copy, "Copying data to the GPU");

    ///////////////////////////////////////////////////////
    gpuTKTime_start(Compute, "Doing the computation on the GPU");
    // TODO: Configure launch parameters and call kernel
   dim3 blockSize(16, 16);
   dim3 gridSize((width+blockSize.x-1)/blockSize.x, (height+blockSize.y-1)/blockSize.y);
   colorToGrayscale_kernel<<<gridSize, blockSize>>>(deviceOutputImageData, deviceInputImageData, width, height);
   hipDeviceSynchronize(); 
   gpuTKTime_stop(Compute, "Doing the computation on the GPU");

    ///////////////////////////////////////////////////////
    gpuTKTime_start(Copy, "Copying data from the GPU");
    // TODO: Copy data from GPU here
    hipMemcpy(output, deviceOutputImageData, width * height * sizeof(float), hipMemcpyDeviceToHost);
    //printf(); 
    gpuTKTime_stop(Copy, "Copying data from the GPU");

    gpuTKTime_stop(GPU, "Doing GPU Computation (memory + compute)");

    // TODO: Free device memory
    hipFree(deviceInputImageData);
    hipFree(deviceOutputImageData);
}
