#include "hip/hip_runtime.h"
#include "conv2d_gpu.h"

__global__ void convolve2D_constant_kernel(
    const float* input, 
    const float* kernel, 
    float* output, 
    int inputWidth, int inputHeight, 
    int kernelSize, int outputWidth, int outputHeight, 
    int stride, int dilation) {

    // TODO: Implement the kernel for constant memory filtering
}

__global__ void convolve2D_shared_kernel(
    const float* input, 
    const float* kernel, 
    float* output, 
    int inputWidth, int inputHeight, 
    int kernelSize, int outputWidth, int outputHeight, 
    int stride, int dilation) {

    // TODO: Implement the kernel for shared memory filtering
}

void convolve2D(
    const float *input,
    const float *kernel,
    float *output,
    unsigned int inputHeight,
    unsigned int inputWidth,
    unsigned int kernelSize,
    unsigned int outputHeight,
    unsigned int outputWidth,
    unsigned int padding,
    unsigned int stride,
    unsigned int dilation) {

    // Allocate device memory
    float *deviceInput, *deviceKernel, *deviceOutput;

    // TODO: Complete host function.
}