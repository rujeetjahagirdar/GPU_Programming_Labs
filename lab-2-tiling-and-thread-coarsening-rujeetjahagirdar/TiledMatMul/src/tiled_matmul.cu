
#include <hip/hip_runtime.h>
// TODO: Insert code to implement tiled matrix multiplication

#define TILE_WIDTH 2
__global__ void MatMulKernel(float* M, float* N, float* P, int Width) {
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;

    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    //int TILE_WIDTH = 32;
    __shared__ float Mds[TILE_WIDTH][TILE_WIDTH];
    __shared__ float Nds[TILE_WIDTH][TILE_WIDTH];

    // Identify the row and column of the P element to work on
    int Row = by * TILE_WIDTH + ty;
    int Col = bx * TILE_WIDTH + tx;

    float Pvalue = 0;
    for (int ph = 0; ph < Width / TILE_WIDTH; ++ph) {
        // Collaborative loading of M and N tiles into shared memory
        Mds[ty][tx] = M[Row * Width + ph * TILE_WIDTH + tx];
        Nds[ty][tx] = N[(ph * TILE_WIDTH + ty) * Width + Col];
        __syncthreads();

        for (int k = 0; k < TILE_WIDTH; ++k) {
            Pvalue += Mds[ty][k] * Nds[k][tx];
        }
        __syncthreads();
    }

    P[Row * Width + Col] = Pvalue;
}

int tiledmm(float *A_h, float *B_h, float *C_h, int numARows, int numACols,
           int numBRows, int numBCols) {
    float *A_d, *B_d, *C_d;
//    gpuTKTime_start(GPU, "Allocating GPU memory.");
    // TODO: Allocate GPU memory here
    // Don't forget to wrap the function calls with gpuTKCheck() macro
    //gpuTKCheck(cudaMalloc(&A_d, numARows * numACols * sizeof(float)));
    //gpuTKCheck(cudaMalloc(&B_d, numBRows * numBCols * sizeof(float)));
    //gpuTKCheck(cudaMalloc(&C_d, numARows * numBCols * sizeof(float)));
    hipMalloc(&A_d, numARows * numACols * sizeof(float));
    hipMalloc(&B_d, numBRows * numBCols * sizeof(float));
    hipMalloc(&C_d, numARows * numBCols * sizeof(float));

//    gpuTKTime_stop(GPU, "Allocating GPU memory.");

  //  gpuTKTime_start(GPU, "Copying input memory to the GPU.");
    // TODO: Copy memory to the GPU here
//    gpuTKCheck(cudaMemcpy(A_d, A_h, numARows * numACols * sizeof(float), cudaMemcpyHostToDevice));
  //  gpuTKCheck(cudaMemcpy(B_d, B_h, numBRows * numBCols * sizeof(float), cudaMemcpyHostToDevice));
    hipMemcpy(A_d, A_h, numARows * numACols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B_h, numBRows * numBCols * sizeof(float), hipMemcpyHostToDevice); 
//gpuTKTime_stop(GPU, "Copying input memory to the GPU.");

    // TODO: Initialize the grid and block dimensions here
//    dim3 blocksperGrid(ceil(numARows/32.0), ceil(numBCols/32.0),1);
    dim3 blocksperGrid(ceil(numBCols/TILE_WIDTH), ceil(numARows/TILE_WIDTH), 1);
    dim3 threadsperBlock(TILE_WIDTH, TILE_WIDTH, 1);

//    gpuTKLog(TRACE, "The block dimensions are ", blockDim.x, " x ", blockDim.y);
//    gpuTKLog(TRACE, "The grid dimensions are ", gridDim.x, " x ", gridDim.y);

    //gpuTKTime_start(Compute, "Performing CUDA computation");
    // TODO: Launch the GPU Kernel here
    MatMulKernel<<<blocksperGrid, threadsperBlock>>>(A_d, B_d, C_d, numACols);

    hipDeviceSynchronize();
    //gpuTKTime_stop(Compute, "Performing CUDA computation");

    //gpuTKTime_start(Copy, "Copying output memory to the CPU");
    // TODO: Copy the GPU memory back to the CPU here
//    gpuTKCheck(cudaMemcpy(C_h, C_d, numARows * numBCols * sizeof(float), cudaMemcpyDeviceToHost));
    hipMemcpy(C_h, C_d, numARows * numBCols * sizeof(float), hipMemcpyDeviceToHost);
    //gpuTKTime_stop(Copy, "Copying output memory to the CPU");

    //gpuTKTime_start(GPU, "Freeing GPU Memory");
    // TODO: Free the GPU memory here
    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);
    //gpuTKTime_stop(GPU, "Freeing GPU Memory");

    return 0;
}
